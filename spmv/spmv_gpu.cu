
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#define CHECK(call)                                                                         \
{                                                                                           \
    const hipError_t err = call;                                                           \
    if(err != hipSuccess) {                                                                \
        printf("%s in %s at line %d\n", hipGetErrorName(err), __FILE__, __LINE__);         \
        exit(EXIT_FAILURE);                                                                 \
    }                                                                                       \
}                                                                                           

#define CHECK_KERNEL_CALL()                                                                 \
{                                                                                           \
    const hipError_t err = hipGetLastError();                                             \
    if(err != hipSuccess) {                                                                \
        printf("%s in %s at line %d\n", hipGetErrorName(err), __FILE__, __LINE__);         \
        exit(EXIT_FAILURE);                                                                 \
    }                                                                                       \
}                                                                                           


double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, const char *filename, int *num_rows, int *num_cols, int *num_vals) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    printf("Got the number of rows etc...\n");
    printf("%d rows, %d cols, %d vals\n", *num_rows, *num_cols, *num_vals);
    int *row_ptr_t = (int *) malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *) malloc(*num_vals * sizeof(int));
    float *values_t = (float *) malloc(*num_vals * sizeof(float));
    
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *) malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++) {
        row_occurances[i] = 0;
    }
    

    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        
        row_occurances[row]++;
    }
    
    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++) {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);
    
    printf("row_ptr array ready\n");

    // Set the file position to the beginning of the file
    rewind(file);
    

    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++) {
        col_ind_t[i] = -1;
    }
    
    printf("col_ind array ready\n");

    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        row--;
        column--;
        
        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1) {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        i = 0;
    }

    printf("values saved in memory\n");
    
    fclose(file);
    

    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;

    printf("Set up was completed!\n\n");
}

// CPU implementation of SPMV using CSR, DO NOT CHANGE THIS
void spmv_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *x, float *y) {
    for (int i = 0; i < num_rows; i++) {
        float dotProduct = 0;
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        
        for (int j = row_start; j < row_end; j++) {
            dotProduct += values[j] * x[col_ind[j]];
        }
        
        y[i] = dotProduct;
    }
}

void check_results(const float *a1, const float *a2, const int rows) {
    for(int i = 0; i < rows; i++) {
        if(abs(a1[i] - a2[i]) > 0.003) {
            printf("should be %lf, got %lf at index %d\n", a1[i], a2[i], i);
            return;
        }
    }
    printf("COMPUTATION WAS CORRECT\n");
}


__global__ void spmv_csr_gpu(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *d_x, float *y, const int num_threads) {
    
    int single_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    for(int index = single_thread_id; index < num_rows; index = index + num_threads) {
        
        float dot_product = 0;
        int row_start = row_ptr[index];
        int row_end = row_ptr[index+1];

        for(int j = row_start; j < row_end; j++) {
            dot_product += values[j] * d_x[col_ind[j]];
        }

        y[index] = dot_product;
    }
}



int main(int argc, const char * argv[]) {

    if (argc != 3) {
        printf("Usage: ./exec matrix_file num_threads");
        return 0;
    }
    
    // Variables for cpu

    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    
    int *d_row_ptr, *d_col_ind;
    float *d_values;
    float *d_x;
    float *d_y;
    
    const char *filename = argv[1];
    const int num_threads = atoi(argv[2]);

    double start_cpu, end_cpu;
    double start_gpu, end_gpu;
    
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    
    float *x = (float *) malloc(num_rows * sizeof(float));
    float *y_sw = (float *) malloc(num_rows * sizeof(float));
    float *result = (float *) malloc(num_rows * sizeof(float));

    // Allocating gpu memory for the various arrays
    printf("Allocating GPU memory\n");    

    CHECK(hipMalloc(&d_row_ptr, (num_rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_col_ind, num_vals * sizeof(int)));
    CHECK(hipMalloc(&d_values, num_vals * sizeof(float)));
    CHECK(hipMalloc(&d_y, (num_rows + 1) * sizeof(float)));

    // Generate a random vector

    srand(time(NULL));

    for (int i = 0; i < num_rows; i++) {
        x[i] = (rand()%100)/(rand()%100+1); //the number we use to divide cannot be 0, that's the reason of the +1
    }
    
    CHECK(hipMalloc(&d_x, (num_rows + 1) * sizeof(float)));

    // Copying data from cpu to gpu

    printf("Copying data from CPU to GPU\n");
    CHECK(hipMemcpy(d_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_col_ind, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_values, values, num_vals * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_x, x, (num_rows + 1) * sizeof(float), hipMemcpyHostToDevice));

    // Compute in GPU

    printf("GPU computation\n");
    dim3 threadsPerBlock(256);
    dim3 numBlocks(num_threads/256);

    start_gpu = get_time();
    spmv_csr_gpu<<<numBlocks, threadsPerBlock>>>(
        d_row_ptr,
        d_col_ind,
        d_values,
        num_rows,
        d_x,
        d_y,
        num_threads
      );
    CHECK_KERNEL_CALL();
    hipDeviceSynchronize();

    CHECK(hipMemcpy(result, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost));

    end_gpu = get_time();
    
    // Compute in sw

    printf("CPU computation\n");

    start_cpu = get_time();
    spmv_csr_sw(row_ptr, col_ind, values, num_rows, x, y_sw);
    end_cpu = get_time();

    // Verifying result
    check_results(y_sw, result, num_rows);

    // Print time
    printf("SPMV Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SPMV Time GPU: %.10lf\n", end_gpu - start_gpu);
    
    // Free    
    free(row_ptr);
    free(col_ind);
    free(values);
    free(y_sw);
    free(result);

    // GPU memory free
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
    
    return 0;
}
